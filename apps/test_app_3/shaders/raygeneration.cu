#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "half_common.h"
#include "random_number_generators.h"
#include "transform.h"

typedef mi::neuraylib::Shading_state_material Mdl_state;

extern "C" __constant__ SystemData sysData;

__forceinline__ __device__ int2 pixel_from_world_coord(const float2 screen, const LensRay ray, float3 world_coord)
{
    const CameraDefinition camera = sysData.cameraDefinitions[0];

    float A[9] = {
        camera.U.x, camera.U.y, camera.U.z,
        camera.V.x, camera.V.y, camera.V.z,
        camera.W.x, camera.W.y, camera.W.z,
    };
    float3 x = world_coord - camera.P;

    //    0  1  2
    // 0  0  3  6
    // 1  1  4  7
    // 2  2  5  8

    float dt = A[0] * (A[4] * A[8] - A[5] * A[7]) -
               A[3] * (A[1] * A[8] - A[7] * A[2]) +
               A[6] * (A[1] * A[5] - A[4] * A[2]);
    float invdet = 1.f / dt;

    float minv[9];
    minv[0] = (A[4] * A[8] - A[5] * A[7]) * invdet;
    minv[3] = (A[6] * A[5] - A[3] * A[8]) * invdet;
    minv[6] = (A[3] * A[7] - A[6] * A[4]) * invdet;
    minv[1] = (A[7] * A[2] - A[1] * A[8]) * invdet;
    minv[4] = (A[0] * A[8] - A[6] * A[2]) * invdet;
    minv[7] = (A[1] * A[6] - A[0] * A[7]) * invdet;
    minv[2] = (A[1] * A[5] - A[2] * A[4]) * invdet;
    minv[5] = (A[2] * A[3] - A[0] * A[5]) * invdet;
    minv[8] = (A[0] * A[4] - A[1] * A[3]) * invdet;

    float nx = x.x * minv[0] + x.y * minv[3] + x.z * minv[6];
    float ny = x.x * minv[1] + x.y * minv[4] + x.z * minv[7];
    float nz = x.x * minv[2] + x.y * minv[5] + x.z * minv[8];
    float2 ndc;
    ndc.x = nx / nz;
    ndc.y = ny / nz;

    float2 fragment = (ndc + 1.0f) * 0.5f * screen;
    int2 pixel_index;
    pixel_index.x = (int)fragment.x;
    pixel_index.y = (int)fragment.y;

    return pixel_index;
}


__forceinline__ __device__ float3 safe_div(const float3& a, const float3& b)
{
  const float x = (b.x != 0.0f) ? a.x / b.x : 0.0f;
  const float y = (b.y != 0.0f) ? a.y / b.y : 0.0f;
  const float z = (b.z != 0.0f) ? a.z / b.z : 0.0f;

  return make_float3(x, y, z);
}

__forceinline__ __device__ float sampleDensity(const float3& albedo,
                                               const float3& throughput,
                                               const float3& sigma_t,
                                               const float   u,
                                               float3&       pdf)
{
  const float3 weights = throughput * albedo;

  const float sum = weights.x + weights.y + weights.z;
  
  pdf = (0.0f < sum) ? weights / sum : make_float3(1.0f / 3.0f);

  if (u < pdf.x)
  {
    return sigma_t.x;
  }
  if (u < pdf.x + pdf.y)
  {
    return sigma_t.y;
  }
  return sigma_t.z;
}

// Determine Henyey-Greenstein phase function cos(theta) of scattering direction
__forceinline__ __device__ float sampleHenyeyGreensteinCos(const float xi, const float g)
{
  // PBRT v3: Chapter 15.2.3
  if (fabsf(g) < 1e-3f) // Isotropic.
  {
    return 1.0f - 2.0f * xi;
  }

  const float s = (1.0f - g * g) / (1.0f - g + 2.0f * g * xi);
  return (1.0f + g * g - s * s) / (2.0f * g);
}

// Determine scatter reflection direction with Henyey-Greenstein phase function.
__forceinline__ __device__ void sampleVolumeScattering(const float2 xi, const float g, float3& dir)
{
  const float cost = sampleHenyeyGreensteinCos(xi.x, g);
  
  float sint = 1.0f - cost * cost;
  sint = (0.0f < sint) ? sqrtf(sint) : 0.0f;
 
  const float phi = 2.0f * M_PIf * xi.y;

  // This vector is oriented in its own local coordinate system:
  const float3 d = make_float3(cosf(phi) * sint, sinf(phi) * sint, cost); 

  // Align the vector with the incoming direction.
  const TBN tbn(dir); // Just some ortho-normal basis along dir as z-axis.
  
  dir = tbn.transformToWorld(d);
}


__forceinline__ __device__ float3 integrator(PerRayData& prd, int index)
{
  // The integrator starts with black radiance and full path throughput.
  prd.radiance   = make_float3(0.0f);
  prd.pdf        = 0.0f;
  prd.throughput = make_float3(1.0f);
  prd.flags      = 0;
  prd.sigma_t    = make_float3(0.0f); // Extinction coefficient: sigma_a + sigma_s.
  prd.walk       = 0;                 // Number of random walk steps taken through volume scattering. 
  prd.eventType  = mi::neuraylib::BSDF_EVENT_ABSORB; // Initialize for exit. (Otherwise miss programs do not work.)
  prd.shadow_ray = false;
  prd.prev_primitive_idx = 0;
  // Nested material handling. 
  prd.idxStack   = 0;
  // Small stack of four entries of which the first is vacuum.
  prd.stack[0].ior     = make_float3(1.0f); // No effective IOR.
  prd.stack[0].sigma_a = make_float3(0.0f); // No volume absorption.
  prd.stack[0].sigma_s = make_float3(0.0f); // No volume scattering.
  prd.stack[0].bias    = 0.0f;              // Isotropic volume scattering.

  // Put payload pointer into two unsigned integers. Actually const, but that's not what optixTrace() expects.
  uint2 payload = splitPointer(&prd);

  // Russian Roulette path termination after a specified number of bounces needs the current depth.
  int depth = 0; // Path segment index. Primary ray is depth == 0.

  while (depth < sysData.pathLengths.y)
  {
      // if (index == 0) {
      //     printf("depth = %d\tsysData.pathLengths = %d, %d\tSPP = %d\n",
      //            depth, sysData.pathLengths.x, sysData.pathLengths.y, sysData.spp);
      // }

    // Self-intersection avoidance:
    // Offset the ray t_min value by sysData.sceneEpsilon when a geometric primitive was hit by the previous ray.
    // Primary rays and volume scattering miss events will not offset the ray t_min.
    const float epsilon = (prd.flags & FLAG_HIT) ? sysData.sceneEpsilon : 0.0f;

    prd.wo       = -prd.wi;        // Direction to observer.
    prd.distance = RT_DEFAULT_MAX; // Shoot the next ray with maximum length.
    prd.flags    = 0;

    // Special cases for volume scattering!
    if (0 < prd.idxStack) // Inside a volume?
    {
      // Note that this only supports homogeneous volumes so far! 
      // No change in sigma_s along the random walk here.
      const float3 sigma_s = prd.stack[prd.idxStack].sigma_s;

      if (isNotNull(sigma_s)) // We're inside a volume and it has volume scattering?
      {
        // Indicate that we're inside a random walk. This changes the behavior of the miss programs.
        prd.flags |= FLAG_VOLUME_SCATTERING;

        // Random walk through scattering volume, sampling the distance.
        // Note that the entry and exit of the volume is done according to the BSDF sampling.
        // Means glass with volume scattering will still do the proper refractions.
        // When the number of random walk steps has been exceeded, the next ray is shot with distance RT_DEFAULT_MAX
        // to hit something. If that results in a transmission the scattering volume is left.
        // If not, this continues until the maximum path length has been exceeded.
        if (prd.walk < sysData.walkLength)
        {
          const float3 albedo = safe_div(sigma_s, prd.sigma_t);
          const float2 xi     = rng2(prd.seed);
          
          const float s = sampleDensity(albedo, prd.throughput, prd.sigma_t, xi.x, prd.pdfVolume);

          // Prevent logf(0.0f) by sampling the inverse range (0.0f, 1.0f].
          prd.distance = -logf(1.0f - xi.y) / s;
        }
      }
    }

#if (USE_SHADER_EXECUTION_REORDERING == 0 || OPTIX_VERSION < 80000)
    // Note that the primary rays and volume scattering miss cases do not offset the ray t_min by sysSceneEpsilon.
    optixTrace(sysData.topObject,
               prd.pos, prd.wi, // origin, direction
               epsilon, prd.distance, 0.0f, // tmin, tmax, time
               OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE, 
               TYPE_RAY_RADIANCE, NUM_RAY_TYPES, TYPE_RAY_RADIANCE,
               payload.x, payload.y);
#else
    // OptiX Shader Execution Reordering (SER) implementation.
    optixTraverse(sysData.topObject,
                  prd.pos, prd.wi, // origin, direction
                  epsilon, prd.distance, 0.0f, // tmin, tmax, time
                  OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE, 
                  TYPE_RAY_RADIANCE, NUM_RAY_TYPES, TYPE_RAY_RADIANCE,
                  payload.x, payload.y);

    unsigned int hint = 0; // miss uses some default value. The record type itself will distinguish this case.
    if (optixHitObjectIsHit())
    {
      const int idMaterial = sysData.geometryInstanceData[optixHitObjectGetInstanceId()].ids.x;
      hint = sysData.materialDefinitionsMDL[idMaterial].indexShader; // Shader configuration only.
    }
    optixReorder(hint, sysData.numBitsShaders);

    optixInvoke(payload.x, payload.y);
#endif

    // Path termination by miss shader or sample() routines.
    if ((prd.eventType == mi::neuraylib::BSDF_EVENT_ABSORB) || isNull(prd.throughput))
    {
      break;
    } 

    // Unbiased Russian Roulette path termination.
    if (sysData.pathLengths.x <= depth) // Start termination after a minimum number of bounces.
    {
      const float probability = fmaxf(prd.throughput);

      if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
      {
        break;
      }

      prd.throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
    }

    // We're inside a volume and the scatter ray missed.
    if (prd.flags & FLAG_VOLUME_SCATTERING_MISS) // This implies FLAG_VOLUME_SCATTERING.
    {
      // Random walk through scattering volume, sampling the direction according to the phase function.
      sampleVolumeScattering(rng2(prd.seed), prd.stack[prd.idxStack].bias, prd.wi);
    }

    ++depth; // Next path segment.
    prd.depth += 1;
  }
  
  return prd.radiance;
}

__forceinline__ __device__ float3 expensive_shadow_ray(Reservoir& rsv) {
    float3 origin  = rsv.nearest_hit;
    float3 dir     = rsv.y.direction;
    float3 last_wi = rsv.last_wi;
    uint32_t primitive_idx = rsv.prev_primitive_idx;

    PerRayData prd;
    prd.pos                = origin;
    prd.wi                 = dir;
    prd.prev_primitive_idx = primitive_idx;
    prd.prev_instance_id  = rsv.prev_instance_id;

    prd.shadow_ray         = true;
    prd.distance           = rsv.y.distance;
    prd.last_barycentrics  = rsv.last_barycentrics;

    prd.last_wi            = last_wi;

    uint2 payload = splitPointer(&prd);

#if (USE_SHADER_EXECUTION_REORDERING == 0 || OPTIX_VERSION < 80000)
    // Note that the primary rays and volume scattering miss cases do not offset the ray t_min by sysSceneEpsilon.
    optixTrace(sysData.topObject,
               prd.pos, prd.wi, // origin, direction
               sysData.sceneEpsilon, prd.distance, 0.0f, // tmin, tmax, time
               OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE,
               TYPE_RAY_RADIANCE, NUM_RAY_TYPES, TYPE_RAY_RADIANCE,
               payload.x, payload.y);
#else
    // OptiX Shader Execution Reordering (SER) implementation.
    optixTraverse(sysData.topObject,
                  prd.pos, prd.wi, // origin, direction
                  sysData.sceneEpsilon, prd.distance+2, 0.0f, // tmin, tmax, time
                  OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE,
                  TYPE_RAY_RADIANCE, NUM_RAY_TYPES, TYPE_RAY_RADIANCE,
                  payload.x, payload.y);

    unsigned int hint = 0; // miss uses some default value. The record type itself will distinguish this case.
    if (optixHitObjectIsHit())
    {
        const int idMaterial = sysData.geometryInstanceData[optixHitObjectGetInstanceId()].ids.x;
        hint = sysData.materialDefinitionsMDL[idMaterial].indexShader; // Shader configuration only.
    }
    optixReorder(hint, sysData.numBitsShaders);

    optixInvoke(payload.x, payload.y);
#endif

    return prd.throughput;

}


__forceinline__ __device__ unsigned int distribute(const uint2 launchIndex)
{
  // First calculate block coordinates of this launch index.
  // That is the launch index divided by the tile dimensions. (No operator>>() on vectors?)
  const unsigned int xBlock = launchIndex.x >> sysData.tileShift.x;
  const unsigned int yBlock = launchIndex.y >> sysData.tileShift.y;
  
  // Each device needs to start at a different column and each row should start with a different device.
  const unsigned int xTile = xBlock * sysData.deviceCount + ((sysData.deviceIndex + yBlock) % sysData.deviceCount);

  // The horizontal pixel coordinate is: tile coordinate * tile width + launch index % tile width.
  return xTile * sysData.tileSize.x + (launchIndex.x & (sysData.tileSize.x - 1)); // tileSize needs to be power-of-two for this modulo operation.
}

extern "C" __global__ void __raygen__path_tracer()
{
#if USE_TIME_VIEW
    clock_t clockBegin = clock();
#endif
    const uint2 theLaunchDim   = make_uint2(optixGetLaunchDimensions()); // For multi-GPU tiling this is (resolution + deviceCount - 1) / deviceCount.
    const uint2 theLaunchIndex = make_uint2(optixGetLaunchIndex());
    
    PerRayData prd;
    
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<4>(theLaunchDim.x * theLaunchIndex.y + theLaunchIndex.x, sysData.iterationIndex); // PERF This template really generates a lot of instructions.
    prd.launchDim = theLaunchDim;
    prd.launchIndex = theLaunchIndex;
    prd.depth = 0;
    
    // Decoupling the pixel coordinates from the screen size will allow for partial rendering algorithms.
    // Resolution is the actual full rendering resolution and for the single GPU strategy, theLaunchDim == resolution.
    const float2 screen = make_float2(sysData.resolution); // == theLaunchDim for rendering strategy RS_SINGLE_GPU.
    const float2 pixel  = make_float2(theLaunchIndex);
    const float2 sample = rng2(prd.seed);
    
    // Lens shaders
    const LensRay ray = optixDirectCall<LensRay, const float2, const float2, const float2>(sysData.typeLens, screen, pixel, sample);
    
    prd.pos = ray.org;
    prd.wi  = ray.dir;
    
    float3 radiance = float3({0.0, 0.0, 0.0});
    
    Reservoir* ris_output_reservoir_buffer = reinterpret_cast<Reservoir*>(sysData.RISOutputReservoirBuffer);
    Reservoir* spatial_output_reservoir_buffer = reinterpret_cast<Reservoir*>(sysData.SpatialOutputReservoirBuffer);
    Reservoir* temp_reservoir_buffer = reinterpret_cast<Reservoir*>(sysData.TempReservoirBuffer);
    
    const unsigned int index = theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x;
    int lidx_ris = (theLaunchDim.x * theLaunchDim.y * sysData.cur_iter) + index;
    int lidx_prev = (theLaunchDim.x * theLaunchDim.y * (sysData.cur_iter - 1)) + index;

    // don't question this too much
    const PaneFlags& pane_flags = sysData.num_panes == 1 ? sysData.pane_a_flags :
                                  sysData.num_panes == 2 ? (theLaunchIndex.x < theLaunchDim.x * 0.5) ?
                                                            sysData.pane_a_flags : sysData.pane_b_flags :
                                  sysData.num_panes == 3 ? (theLaunchIndex.x < theLaunchDim.x * 0.33) ?
                                                            sysData.pane_a_flags : (theLaunchIndex.x < theLaunchDim.x * 0.67) ?
                                                            sysData.pane_b_flags : sysData.pane_c_flags
                                                            : sysData.pane_c_flags;

    if (pane_flags.do_reference) {
        prd.do_reference           = true;
        prd.num_ris_samples        = 0;
        prd.do_temporal_resampling = false;
        prd.do_spatial_resampling  = false;
    } else {
        prd.do_reference           = false;
        prd.num_ris_samples        = pane_flags.ris_samples;
        prd.do_temporal_resampling = pane_flags.do_temporal_reuse;
        prd.do_spatial_resampling  = pane_flags.do_spatial_reuse;
    }
    
    prd.launch_linear_index = lidx_ris;

    // ########################
    // HANDLE RIS LOGIC
    // ########################
    if(sysData.cur_iter != sysData.spp) {
        if (prd.num_ris_samples > 0) {
            ris_output_reservoir_buffer[lidx_ris] = Reservoir({0, 0, 0, 0});
        }
        radiance = integrator(prd, index);

        // integrator(prd, index);
    }

    if (index == 131328 && sysData.cur_iter == 0) {
        printf("\n\n\n");
    }

    // ########################
    //  HANDLE TEMPORAL LOGIC
    // ########################
    if (prd.do_temporal_resampling && !sysData.first_frame && sysData.cur_iter != sysData.spp) {
        Reservoir &current_pixel_prev_reservoir = temp_reservoir_buffer[lidx_ris]; // get current pixel's previous reservoir
        Reservoir &current_reservoir = ris_output_reservoir_buffer[lidx_ris];        // choose current reservoir
        LightSample& y1 = current_reservoir.y;

        if (index == 131328) {
            printf("\nrunning temporal reuse: sysData.cur_iter = %d\n", sysData.cur_iter);
            printf("Cur pixel prev temp reservoir w_sum= %f\tW = %f\tM = %d\n",
                   current_pixel_prev_reservoir.w_sum, current_pixel_prev_reservoir.W, current_pixel_prev_reservoir.M);
            printf("CUR RIS reservoir initial w_sum= %f\tW = %f\tM = %d\n", current_reservoir.w_sum, current_reservoir.W, current_reservoir.M);
            printf("\tCUR direction = %f,%f,%f\n", y1.direction.x, y1.direction.y, y1.direction.z);
            //printf("\tCUR light pdf = %f\tphat = %f\n", y1.pdf, length(y1.radiance_over_pdf));
            printf("\tCUR length(final_reservoir.throughput_x_bxdf) = %f\n", length(current_reservoir.throughput_x_bxdf));

        }

        // updateReservoir(
        //     &s,
        //     &y1,
        //     length(y1.radiance_over_pdf) * y1.pdf * current_reservoir.W * current_reservoir.M,
        //     &prd.seed
        // );
        int2 current_pixel_prev_coord = pixel_from_world_coord(screen, ray, current_pixel_prev_reservoir.nearest_hit);
        int2 current_pixel_curr_coord = pixel_from_world_coord(screen, ray, current_reservoir.nearest_hit);
        int offset_x = theLaunchIndex.x - current_pixel_prev_coord.x;
        int offset_y = theLaunchIndex.y - current_pixel_prev_coord.y;

        if (index == 131328) {
            printf("current_pixel_prev_coord %d, %d\ncurrent_pixel_curr_coord %d, %d\n",
                   current_pixel_prev_coord.x, current_pixel_prev_coord.y, current_pixel_curr_coord.x, current_pixel_curr_coord.y);
            printf("theLaunchIndex %d, %d\n",
                   theLaunchIndex.x, theLaunchIndex.y);
            printf("offset = %d %d\n", offset_x, offset_y);
        }

        int prev_coord_x = theLaunchIndex.x + offset_x;
        int prev_coord_y = theLaunchIndex.y + offset_y;

        bool prev_coord_offscreen = false;
        if (prev_coord_x < 0 || prev_coord_y < 0)
            prev_coord_offscreen = true;
        else if (prev_coord_x >= theLaunchDim.x || prev_coord_y >= theLaunchDim.y)
            prev_coord_offscreen = true;

        bool prev_coord_did_hit = true;
        if (
            current_reservoir.nearest_hit.x == 0.f &&
            current_reservoir.nearest_hit.y == 0.f &&
            current_reservoir.nearest_hit.z == 0.f
        ){
            prev_coord_did_hit = false;
        }

        bool prev_too_far = sqrt((double)(offset_x * offset_x + offset_y * offset_y)) > 10.0;

        if (!prev_coord_offscreen && prev_coord_did_hit && !prev_too_far) {
            // select previous frame's reservoir and combine it
            // and only combine if you actually hit something (empty reservoir bad!)
            int prev_index =
                theLaunchDim.x * theLaunchDim.y * (sysData.cur_iter) +
                prev_coord_y * theLaunchDim.x + prev_coord_x; // TODO: how to calculate motion vector??

            Reservoir& prev_frame_reservoir = temp_reservoir_buffer[prev_index];
            LightSample& y2 = prev_frame_reservoir.y;


            if (index == 131328) {
                printf("Prev frame reservoir initial w_sum= %f\tW = %f\tM = %d\n",
                       prev_frame_reservoir.w_sum, prev_frame_reservoir.W, prev_frame_reservoir.M);
                printf("PREV FRAME: idx %d reservoir w_sum = %f\tW = %f\tM = %d\n", prev_index, prev_frame_reservoir.w_sum, prev_frame_reservoir.W, prev_frame_reservoir.M);
                printf("\tPREV direction = %f,%f,%f\n", y2.direction.x, y2.direction.y, y2.direction.z);
                //printf("\tPREV light pdf = %f\tphat = %f\n", y2.pdf, length(y2.radiance_over_pdf));
                printf("\tPREV length(final_reservoir.throughput_x_bxdf) = %f\n", length(prev_frame_reservoir.throughput_x_bxdf));
            }

            // if (prev_frame_reservoir.M >= current_reservoir.M){
            //     prev_frame_reservoir.M = current_reservoir.M;
            // }

            float prv_phat = length(y2.radiance_over_pdf) * y2.pdf;
            float cur_phat = length(y1.radiance_over_pdf) * y1.pdf;

            float dist_between_hits = length(prev_frame_reservoir.nearest_hit - current_reservoir.nearest_hit);
            float wght_due_to_dist  = 1.f / (dist_between_hits + 1.f);

            float m_prev = balanceHeuristic(prev_frame_reservoir.M * prv_phat, current_reservoir.M * cur_phat) * wght_due_to_dist;
            if (prev_frame_reservoir.W > 0) {
                updateReservoir(
                    &current_reservoir,
                    &y2,
                    length(y2.radiance_over_pdf) * y2.pdf * prev_frame_reservoir.W * m_prev,
                    &prd.seed
                );
            }

            if (index == 131328) {
                printf("Temporal reuse after combination w_sum= %f\tW = %f\tM = %d\n\n",
                       current_reservoir.w_sum, current_reservoir.W, current_reservoir.M);
            }
            current_reservoir.M = min(current_reservoir.M + prev_frame_reservoir.M, 40);
            current_reservoir.W =
                (1.0f / (length(y1.radiance_over_pdf) * y1.pdf)) * // 1 / p_hat
                current_reservoir.w_sum;

            if (isnan(current_reservoir.W) || current_reservoir.M == 0.f) clear_reservoir(current_reservoir);

            if (index == 131328) {
                printf("Temporal reuse end result w_sum= %f\tW = %f\tM = %d\n\n",
                       current_reservoir.w_sum, current_reservoir.W, current_reservoir.M);
            }
        }
    }


    // ########################
    // HANDLE SPATIAL LOGIC
    // ########################
    if (prd.do_spatial_resampling && sysData.cur_iter != sysData.spp) {
        if (sysData.cur_iter == 0) {
            // No spatial reuse, simply pass the current samples forward
            spatial_output_reservoir_buffer[lidx_ris] = ris_output_reservoir_buffer[lidx_ris];;
        } else {
            if (index == 131328) {
                printf("running spatial reuse: %d\t sysData.cur_iter = %d\n", prd.do_spatial_resampling,  sysData.cur_iter);
            }
            Reservoir& updated_reservoir = ris_output_reservoir_buffer[lidx_ris];
            if (index == 131328) {
                printf("spatial reservoir TEST INTIIAL VALUE w_sum = %f\tW = %f\tM = %d\n", updated_reservoir.w_sum, updated_reservoir.W, updated_reservoir.M);
                printf("\tlight sample direction = %f,%f,%f\n", updated_reservoir.y.direction.x, updated_reservoir.y.direction.y,updated_reservoir.y.direction.z);

            }

            int k = 5;
            int radius = 30;
            int num_k_sampled = 0;
            int total_M = updated_reservoir.M;

            while(num_k_sampled < k){
                float2 sample = (rng2(prd.seed) - 0.5f) * radius * 2.0f;
                float squared_dist = sample.x * sample.x + sample.y * sample.y;
                if(squared_dist > radius * radius) continue;

                int _x = (int)sample.x + theLaunchIndex.x;
                int _y = (int)sample.y + theLaunchIndex.y;
                if(_x < 0 || _x >= theLaunchDim.x) continue;
                if(_y < 0 || _y >= theLaunchDim.y) continue;
                if(_x == theLaunchIndex.x && _y == theLaunchIndex.y) continue;

                num_k_sampled += 1;

                unsigned int neighbor_index =
                    theLaunchDim.x * theLaunchDim.y * (sysData.cur_iter - 1) +
                    _y * theLaunchDim.x + _x;
                Reservoir* neighbor_reservoir = &spatial_output_reservoir_buffer[neighbor_index];
                LightSample* y = &neighbor_reservoir->y;

                if (index == 131328) {
                    printf("NEIGHBOR: %d reservoir w_sum = %f\tW = %f\tM = %d\n", neighbor_index, neighbor_reservoir->w_sum, neighbor_reservoir->W, neighbor_reservoir->M);
                    printf("\tneighbor direction = %f,%f,%f\n", y->direction.x, y->direction.y, y->direction.z);
                    printf("\tneighbor light pdf = %f\tphat = %f\n", y->pdf, length(y->radiance_over_pdf));

                }

                float nbr_phat = length(y->radiance_over_pdf) * y->pdf;
                float cur_phat = length(updated_reservoir.y.radiance_over_pdf) * updated_reservoir.y.pdf;

                float dist_between_hits = length(neighbor_reservoir->nearest_hit - updated_reservoir.nearest_hit);
                float wght_due_to_dist  = 1.f / (dist_between_hits + 1.f);

                float m_neighbor = balanceHeuristic(neighbor_reservoir->M * nbr_phat, updated_reservoir.M * cur_phat) * wght_due_to_dist;


                if (neighbor_reservoir->W > 0) {
                    updateReservoir(
                        &updated_reservoir,
                        y,
                        length(y->radiance_over_pdf) * y->pdf * m_neighbor * neighbor_reservoir->W,
                        &prd.seed
                        );
                    total_M += neighbor_reservoir->M;
                }
            }

            LightSample& y = updated_reservoir.y;
            updated_reservoir.M = min(total_M, 40);
            updated_reservoir.W =
                (1.0f / (length(y.radiance_over_pdf) * y.pdf)) * // 1 / p_hat
                updated_reservoir.w_sum;

            if (isnan(updated_reservoir.W) || updated_reservoir.M == 0.f) clear_reservoir(updated_reservoir);

            // Keep a copy of the updated reservoir for the next-frame's temporal reuse
            spatial_output_reservoir_buffer[lidx_ris] = updated_reservoir;
            //radiance = y.f_actual * updated_reservoir.W;
        }
    }


    ////////////////////////////////
    // shoot direct lighting ray
    ////////////////////////////////
    if (prd.num_ris_samples > 0 && sysData.cur_iter != sysData.spp) {
        Reservoir& final_reservoir = ris_output_reservoir_buffer[lidx_ris];
        if (index == 131328) {
            printf("FINAL RESERVOIR w_sum = %f\tW = %f\tM = %d\n", final_reservoir.w_sum, final_reservoir.W, final_reservoir.M);
            printf("\t length(final_reservoir.throughput_x_bxdf) = %f\n", length(final_reservoir.throughput_x_bxdf));

        }
        if (final_reservoir.M > 0 && length(final_reservoir.throughput_x_bxdf) > 0) {
            LightSample& lightSample = final_reservoir.y;
            // Pass the current payload registers through to the shadow ray.
            uint2 payload = splitPointer(&prd);

            prd.flags &= ~FLAG_SHADOW;                  // Clear the shadow flag.

            int tidx = prd.launchIndex.y * prd.launchDim.x + prd.launchIndex.x;
            if (tidx == 131328) {
                printf("about to shoot shadow ray: thePrd.pos = %f,%f,%f, lightSample.direction = %f,%f,%f\n",
                       prd.pos.x,prd.pos.y,prd.pos.z, lightSample.direction.x, lightSample.direction.y, lightSample.direction.z);
                printf("prev_info_valid %d\n", final_reservoir.prev_info_valid);
            }

            // Note that the sysData.sceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval
            // to prevent self-intersections with the actual light geometry in the scene.
            // if (final_reservoir.prev_info_valid) {
            //     if (tidx == 131328) {
            //         printf("about to shoot expensive ray. length(final_reservoir.throughput_x_bxdf) = %f\n", length(final_reservoir.throughput_x_bxdf));
            //     }
            //     final_reservoir.throughput_x_bxdf = expensive_shadow_ray(final_reservoir);
            //     if (tidx == 131328) {
            //         printf("resulting length(final_reservoir.throughput_x_bxdf) = %f\n", length(final_reservoir.throughput_x_bxdf));
            //     }
            // } else {
                optixTrace(sysData.topObject,
                           final_reservoir.nearest_hit, lightSample.direction, // origin, direction
                           sysData.sceneEpsilon, lightSample.distance - sysData.sceneEpsilon, 0.0f, // tmin, tmax, time
                           OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT, // The shadow ray type only uses anyhit programs.
                           TYPE_RAY_SHADOW, NUM_RAY_TYPES, TYPE_RAY_SHADOW,
                           payload.x, payload.y); // Pass through thePrd to the shadow ray.
            //}

            if ((prd.flags & FLAG_SHADOW) == 0) // Visibility test succeeded
            {
                float W = final_reservoir.W;
                float3 f_q =
                    lightSample.pdf * lightSample.radiance_over_pdf *
                    final_reservoir.throughput_x_bxdf * sysData.numLights;

                int tidx = prd.launchIndex.y * prd.launchDim.x + prd.launchIndex.x;
                if (tidx == 131328) {
                    printf("Point NOT in shadow: reservoir w_sum = %f\tW = %f\tM = %d\n", final_reservoir.w_sum, final_reservoir.W, final_reservoir.M);
                }
                radiance += f_q * W;
            } else {
                int tidx = prd.launchIndex.y * prd.launchDim.x + prd.launchIndex.x;
                if (tidx == 131328) {
                    printf("Zeroing out reservoir due to (prd.flags & FLAG_SHADOW) == 0 being false\n");
                }
                clear_reservoir(final_reservoir);
                spatial_output_reservoir_buffer[lidx_ris] = final_reservoir;
                ris_output_reservoir_buffer[lidx_ris] = final_reservoir;
            }
        } else {
            clear_reservoir(final_reservoir);
            spatial_output_reservoir_buffer[lidx_ris] = final_reservoir;
            ris_output_reservoir_buffer[lidx_ris] = final_reservoir;
        }
    }

    if (prd.num_ris_samples > 0 && sysData.cur_iter != 0) {
        // Forward data to the next frame
        temp_reservoir_buffer[lidx_prev] = ris_output_reservoir_buffer[lidx_prev];
    }


#if USE_DEBUG_EXCEPTIONS
    // DEBUG Highlight numerical errors.
    if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
    {
        radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
    }
    else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
    {
        radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
    }
    else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
    {
        radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
    }
#else
    // NaN values will never go away. Filter them out before they can arrive in the output buffer.
    // This only has an effect if the debug coloring above is off!
    if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
    {

#if USE_FP32_OUTPUT
        
        float4* buffer = reinterpret_cast<float4*>(sysData.outputBuffer);

#if USE_TIME_VIEW
        clock_t clockEnd = clock();
        const float alpha = (clockEnd - clockBegin) * sysData.clockScale;
        
        float4 result = make_float4(radiance, alpha);
        
        if (0 < sysData.cur_iter)
        {
            const float4 dst = buffer[index]; // RGBA32F
            
            result = lerp(dst, result, 1.0f / float(sysData.cur_iter + 1)); // Accumulate the alpha as well.
        }
        buffer[index] = result;
#else // if !USE_TIME_VIEW
        if (sysData.iterationIndex < sysData.spp) { // FIXME
            if (0 < sysData.iterationIndex)
            {
                const float4 dst = buffer[index]; // RGBA32F

                radiance = lerp(make_float3(dst), radiance, 1.0f / float(sysData.iterationIndex + 1)); // Only accumulate the radiance, alpha stays 1.0f.
            }
            buffer[index] = make_float4(radiance, 1.0f);
        }
#endif // USE_TIME_VIEW

#else // if !USE_FP32_OUPUT
        
        Half4* buffer = reinterpret_cast<Half4*>(sysData.outputBuffer);

#if USE_TIME_VIEW
        clock_t clockEnd = clock();
        float alpha = (clockEnd - clockBegin) * sysData.clockScale;
        
        if (0 < sysData.cur_iter)
        {
            const float t = 1.0f / float(sysData.cur_iter + 1);
            
            const Half4 dst = buffer[index]; // RGBA16F
            
            radiance.x = lerp(__half2float(dst.x), radiance.x, t);
            radiance.y = lerp(__half2float(dst.y), radiance.y, t);
            radiance.z = lerp(__half2float(dst.z), radiance.z, t);
            alpha      = lerp(__half2float(dst.z), alpha,      t);
        }
        buffer[index] = make_Half4(radiance, alpha);
#else // if !USE_TIME_VIEW
        if (0 < sysData.cur_iter)
        {
            const float t = 1.0f / float(sysData.cur_iter + 1);
            
            const Half4 dst = buffer[index]; // RGBA16F
            
            radiance.x = lerp(__half2float(dst.x), radiance.x, t);
            radiance.y = lerp(__half2float(dst.y), radiance.y, t);
            radiance.z = lerp(__half2float(dst.z), radiance.z, t);
        }
        buffer[index] = make_Half4(radiance, 1.0f);
#endif // USE_TIME_VIEW

#endif // USE_FP32_OUTPUT
    }
}

