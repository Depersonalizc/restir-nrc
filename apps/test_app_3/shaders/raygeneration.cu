#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "half_common.h"
#include "random_number_generators.h"

extern "C" __constant__ SystemData sysData;

__forceinline__ __device__ void matrixMul4x4(float *a, float *b, float *c)
{
    c[0] = a[0] * b[0] + a[4] * b[1] + a[8] * b[2] + a[12] * b[3];
    c[1] = a[1] * b[0] + a[5] * b[1] + a[9] * b[2] + a[13] * b[3];
    c[2] = a[2] * b[0] + a[6] * b[1] + a[10] * b[2] + a[14] * b[3];
    c[3] = a[3] * b[0] + a[7] * b[1] + a[11] * b[2] + a[15] * b[3];

    // col 2
    c[4] = a[0] * b[4] + a[4] * b[5] + a[8] * b[6] + a[12] * b[7];
    c[5] = a[1] * b[4] + a[5] * b[5] + a[9] * b[6] + a[13] * b[7];
    c[6] = a[2] * b[4] + a[6] * b[5] + a[10] * b[6] + a[14] * b[7];
    c[7] = a[3] * b[4] + a[7] * b[5] + a[11] * b[6] + a[15] * b[7];

    // col 3
    c[8] = a[0] * b[8] + a[4] * b[9] + a[8] * b[10] + a[12] * b[11];
    c[9] = a[1] * b[8] + a[5] * b[9] + a[9] * b[10] + a[13] * b[11];
    c[10] = a[2] * b[8] + a[6] * b[9] + a[10] * b[10] + a[14] * b[11];
    c[11] = a[3] * b[8] + a[7] * b[9] + a[11] * b[10] + a[15] * b[11];

    // col 4
    c[12] = a[0] * b[12] + a[4] * b[13] + a[8] * b[14] + a[12] * b[15];
    c[13] = a[1] * b[12] + a[5] * b[13] + a[9] * b[14] + a[13] * b[15];
    c[14] = a[2] * b[12] + a[6] * b[13] + a[10] * b[14] + a[14] * b[15];
    c[15] = a[3] * b[12] + a[7] * b[13] + a[11] * b[14] + a[15] * b[15];
}

__forceinline__ __device__ void matrixMul4x4Transpose(float *a, float *b)
{
    //  0  4  8 12
    //  1  5  9 13
    //  2  6 10 14
    //  3  7 11 15

    b[0] = a[0];
    b[1] = a[4];
    b[2] = a[8];
    b[3] = a[12];

    // col 2
    b[4] = a[1];
    b[5] = a[5];
    b[6] = a[9];
    b[7] = a[13];

    // col 3
    b[8] = a[2];
    b[9] = a[6];
    b[10] = a[10];
    b[11] = a[14];

    // col 4
    b[12] = a[3];
    b[13] = a[7];
    b[14] = a[11];
    b[15] = a[15];
}

__forceinline__ __device__ void matrixVectorMul4x4(float *A, float *b, float *c)
{
    c[0] = A[0] * b[0] + A[4] * b[1] + A[8] * b[2] + A[12] * b[3];
    c[1] = A[1] * b[0] + A[5] * b[1] + A[9] * b[2] + A[13] * b[3];
    c[2] = A[2] * b[0] + A[6] * b[1] + A[10] * b[2] + A[14] * b[3];
    c[3] = A[3] * b[0] + A[7] * b[1] + A[11] * b[2] + A[15] * b[3];
}

__forceinline__ __device__ int2 pixel_from_world_coord(const float2 screen, const LensRay ray, float3 world_coord)
{
    const CameraDefinition camera = sysData.cameraDefinitions[0];

    float A[9] = {
        camera.U.x,
        camera.U.y,
        camera.U.z,
        camera.V.x,
        camera.V.y,
        camera.V.z,
        camera.W.x,
        camera.W.y,
        camera.W.z,
    };
    float3 x = world_coord - camera.P;

    //    0  1  2
    // 0  0  3  6
    // 1  1  4  7
    // 2  2  5  8

    float dt = A[0] * (A[4] * A[8] - A[5] * A[7]) -
               A[3] * (A[1] * A[8] - A[7] * A[2]) +
               A[6] * (A[1] * A[5] - A[4] * A[2]);
    float invdet = 1.f / dt;

    float minv[9];
    minv[0] = (A[4] * A[8] - A[5] * A[7]) * invdet;
    minv[3] = (A[6] * A[5] - A[3] * A[8]) * invdet;
    minv[6] = (A[3] * A[7] - A[6] * A[4]) * invdet;
    minv[1] = (A[7] * A[2] - A[1] * A[8]) * invdet;
    minv[4] = (A[0] * A[8] - A[6] * A[2]) * invdet;
    minv[7] = (A[1] * A[6] - A[0] * A[7]) * invdet;
    minv[2] = (A[1] * A[5] - A[2] * A[4]) * invdet;
    minv[5] = (A[2] * A[3] - A[0] * A[5]) * invdet;
    minv[8] = (A[0] * A[4] - A[1] * A[3]) * invdet;

    float nx = x.x * minv[0] + x.y * minv[3] + x.z * minv[6];
    float ny = x.x * minv[1] + x.y * minv[4] + x.z * minv[7];
    float nz = x.x * minv[2] + x.y * minv[5] + x.z * minv[8];
    float2 ndc;
    ndc.x = nx / nz;
    ndc.y = ny / nz;

    float2 fragment = (ndc + 1.0f) * 0.5f * screen;
    int2 pixel_index;
    pixel_index.x = (int)fragment.x;
    pixel_index.y = (int)fragment.y;

    return pixel_index;
}

__forceinline__ __device__ float3 safe_div(const float3 &a, const float3 &b)
{
    const float x = (b.x != 0.0f) ? a.x / b.x : 0.0f;
    const float y = (b.y != 0.0f) ? a.y / b.y : 0.0f;
    const float z = (b.z != 0.0f) ? a.z / b.z : 0.0f;

    return make_float3(x, y, z);
}

__forceinline__ __device__ float sampleDensity(const float3 &albedo,
                                               const float3 &throughput,
                                               const float3 &sigma_t,
                                               const float u,
                                               float3 &pdf)
{
    const float3 weights = throughput * albedo;

    const float sum = weights.x + weights.y + weights.z;

    pdf = (0.0f < sum) ? weights / sum : make_float3(1.0f / 3.0f);

    if (u < pdf.x)
    {
        return sigma_t.x;
    }
    if (u < pdf.x + pdf.y)
    {
        return sigma_t.y;
    }
    return sigma_t.z;
}

// Determine Henyey-Greenstein phase function cos(theta) of scattering direction
__forceinline__ __device__ float sampleHenyeyGreensteinCos(const float xi, const float g)
{
    // PBRT v3: Chapter 15.2.3
    if (fabsf(g) < 1e-3f) // Isotropic.
    {
        return 1.0f - 2.0f * xi;
    }

    const float s = (1.0f - g * g) / (1.0f - g + 2.0f * g * xi);
    return (1.0f + g * g - s * s) / (2.0f * g);
}

// Determine scatter reflection direction with Henyey-Greenstein phase function.
__forceinline__ __device__ void sampleVolumeScattering(const float2 xi, const float g, float3 &dir)
{
    const float cost = sampleHenyeyGreensteinCos(xi.x, g);

    float sint = 1.0f - cost * cost;
    sint = (0.0f < sint) ? sqrtf(sint) : 0.0f;

    const float phi = 2.0f * M_PIf * xi.y;

    // This vector is oriented in its own local coordinate system:
    const float3 d = make_float3(cosf(phi) * sint, sinf(phi) * sint, cost);

    // Align the vector with the incoming direction.
    const TBN tbn(dir); // Just some ortho-normal basis along dir as z-axis.

    dir = tbn.transformToWorld(d);
}

__forceinline__ __device__ float3 integrator(PerRayData &prd, int index)
{
    // The integrator starts with black radiance and full path throughput.
    prd.radiance = make_float3(0.0f);
    prd.radiance_first_hit = make_float3(0.0f);
    prd.pdf = 0.0f;
    prd.throughput = make_float3(1.0f);
    prd.flags = 0;
    prd.sigma_t = make_float3(0.0f);                  // Extinction coefficient: sigma_a + sigma_s.
    prd.walk = 0;                                     // Number of random walk steps taken through volume scattering.
    prd.eventType = mi::neuraylib::BSDF_EVENT_ABSORB; // Initialize for exit. (Otherwise miss programs do not work.)
    // Nested material handling.
    prd.idxStack = 0;
    // Small stack of four entries of which the first is vacuum.
    prd.stack[0].ior = make_float3(1.0f);     // No effective IOR.
    prd.stack[0].sigma_a = make_float3(0.0f); // No volume absorption.
    prd.stack[0].sigma_s = make_float3(0.0f); // No volume scattering.
    prd.stack[0].bias = 0.0f;                 // Isotropic volume scattering.

    // Put payload pointer into two unsigned integers. Actually const, but that's not what optixTrace() expects.
    uint2 payload = splitPointer(&prd);

    // Russian Roulette path termination after a specified number of bounces needs the current depth.
    int depth = 0; // Path segment index. Primary ray is depth == 0.
    prd.first_hit = true;

    // while (depth < sysData.pathLengths.y)
    while(depth < 1)
    {
        // if (index == 0) {
        //     printf("depth = %d\tsysData.pathLengths = %d, %d\tSPP = %d\n",
        //            depth, sysData.pathLengths.x, sysData.pathLengths.y, sysData.spp);
        // }

        // Self-intersection avoidance:
        // Offset the ray t_min value by sysData.sceneEpsilon when a geometric primitive was hit by the previous ray.
        // Primary rays and volume scattering miss events will not offset the ray t_min.
        const float epsilon = (prd.flags & FLAG_HIT) ? sysData.sceneEpsilon : 0.0f;

        prd.wo = -prd.wi;              // Direction to observer.
        prd.distance = RT_DEFAULT_MAX; // Shoot the next ray with maximum length.
        prd.flags = 0;

        // Special cases for volume scattering!
        if (0 < prd.idxStack) // Inside a volume?
        {
            // Note that this only supports homogeneous volumes so far!
            // No change in sigma_s along the random walk here.
            const float3 sigma_s = prd.stack[prd.idxStack].sigma_s;

            if (isNotNull(sigma_s)) // We're inside a volume and it has volume scattering?
            {
                // Indicate that we're inside a random walk. This changes the behavior of the miss programs.
                prd.flags |= FLAG_VOLUME_SCATTERING;

                // Random walk through scattering volume, sampling the distance.
                // Note that the entry and exit of the volume is done according to the BSDF sampling.
                // Means glass with volume scattering will still do the proper refractions.
                // When the number of random walk steps has been exceeded, the next ray is shot with distance RT_DEFAULT_MAX
                // to hit something. If that results in a transmission the scattering volume is left.
                // If not, this continues until the maximum path length has been exceeded.
                if (prd.walk < sysData.walkLength)
                {
                    const float3 albedo = safe_div(sigma_s, prd.sigma_t);
                    const float2 xi = rng2(prd.seed);

                    const float s = sampleDensity(albedo, prd.throughput, prd.sigma_t, xi.x, prd.pdfVolume);

                    // Prevent logf(0.0f) by sampling the inverse range (0.0f, 1.0f].
                    prd.distance = -logf(1.0f - xi.y) / s;
                }
            }
        }

#if (USE_SHADER_EXECUTION_REORDERING == 0 || OPTIX_VERSION < 80000)
        // Note that the primary rays and volume scattering miss cases do not offset the ray t_min by sysSceneEpsilon.
        optixTrace(sysData.topObject,
                   prd.pos, prd.wi,             // origin, direction
                   epsilon, prd.distance, 0.0f, // tmin, tmax, time
                   OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE,
                   TYPE_RAY_RADIANCE, NUM_RAY_TYPES, TYPE_RAY_RADIANCE,
                   payload.x, payload.y);
#else
        // OptiX Shader Execution Reordering (SER) implementation.
        optixTraverse(sysData.topObject,
                      prd.pos, prd.wi,             // origin, direction
                      epsilon, prd.distance, 0.0f, // tmin, tmax, time
                      OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE,
                      TYPE_RAY_RADIANCE, NUM_RAY_TYPES, TYPE_RAY_RADIANCE,
                      payload.x, payload.y);

        unsigned int hint = 0; // miss uses some default value. The record type itself will distinguish this case.
        if (optixHitObjectIsHit())
        {
            const int idMaterial = sysData.geometryInstanceData[optixHitObjectGetInstanceId()].ids.x;
            hint = sysData.materialDefinitionsMDL[idMaterial].indexShader; // Shader configuration only.
        }
        optixReorder(hint, sysData.numBitsShaders);

        optixInvoke(payload.x, payload.y);
#endif

        // Path termination by miss shader or sample() routines.
        if ((prd.eventType == mi::neuraylib::BSDF_EVENT_ABSORB) || isNull(prd.throughput))
        {
            break;
        }

        // Unbiased Russian Roulette path termination.
        if (sysData.pathLengths.x <= depth) // Start termination after a minimum number of bounces.
        {
            const float probability = fmaxf(prd.throughput);

            if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
            {
                break;
            }

            prd.throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
        }

        // We're inside a volume and the scatter ray missed.
        if (prd.flags & FLAG_VOLUME_SCATTERING_MISS) // This implies FLAG_VOLUME_SCATTERING.
        {
            // Random walk through scattering volume, sampling the direction according to the phase function.
            sampleVolumeScattering(rng2(prd.seed), prd.stack[prd.idxStack].bias, prd.wi);
        }

        ++depth; // Next path segment.
        prd.first_hit = false;
    }

    return prd.radiance;
}

__forceinline__ __device__ unsigned int distribute(const uint2 launchIndex)
{
    // First calculate block coordinates of this launch index.
    // That is the launch index divided by the tile dimensions. (No operator>>() on vectors?)
    const unsigned int xBlock = launchIndex.x >> sysData.tileShift.x;
    const unsigned int yBlock = launchIndex.y >> sysData.tileShift.y;

    // Each device needs to start at a different column and each row should start with a different device.
    const unsigned int xTile = xBlock * sysData.deviceCount + ((sysData.deviceIndex + yBlock) % sysData.deviceCount);

    // The horizontal pixel coordinate is: tile coordinate * tile width + launch index % tile width.
    return xTile * sysData.tileSize.x + (launchIndex.x & (sysData.tileSize.x - 1)); // tileSize needs to be power-of-two for this modulo operation.
}

extern "C" __global__ void __raygen__path_tracer()
{
#if USE_TIME_VIEW
    clock_t clockBegin = clock();
#endif
    const uint2 theLaunchDim = make_uint2(optixGetLaunchDimensions()); // For multi-GPU tiling this is (resolution + deviceCount - 1) / deviceCount.
    const uint2 theLaunchIndex = make_uint2(optixGetLaunchIndex());

    PerRayData prd;

    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<4>(theLaunchDim.x * theLaunchIndex.y + theLaunchIndex.x, sysData.iterationIndex + sysData.rand_seed); // PERF This template really generates a lot of instructions.
    prd.launchDim = theLaunchDim;
    prd.launchIndex = theLaunchIndex;

    // Decoupling the pixel coordinates from the screen size will allow for partial rendering algorithms.
    // Resolution is the actual full rendering resolution and for the single GPU strategy, theLaunchDim == resolution.
    const float2 screen = make_float2(sysData.resolution); // == theLaunchDim for rendering strategy RS_SINGLE_GPU.
    const float2 pixel = make_float2(theLaunchIndex);
    const float2 sample = rng2(prd.seed);

    // Lens shaders
    const LensRay ray = optixDirectCall<LensRay, const float2, const float2, const float2>(sysData.typeLens, screen, pixel, sample);

    prd.pos = ray.org;
    prd.wi = ray.dir;
    float3 radiance = float3({0.0, 0.0, 0.0});

    Reservoir *ris_output_reservoir_buffer = reinterpret_cast<Reservoir *>(sysData.RISOutputReservoirBuffer);
    Reservoir *spatial_output_reservoir_buffer = reinterpret_cast<Reservoir *>(sysData.SpatialOutputReservoirBuffer);
    Reservoir *temp_reservoir_buffer = reinterpret_cast<Reservoir *>(sysData.TempReservoirBuffer);

    const unsigned int index = theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x;
    int lidx_ris = (theLaunchDim.x * theLaunchDim.y * sysData.cur_iter) + index;
    int lidx_spatial = (theLaunchDim.x * theLaunchDim.y * (sysData.cur_iter - 1)) + index;

    // don't question this too much
    const PaneFlags &pane_flags = sysData.num_panes == 1 ? sysData.pane_a_flags : sysData.num_panes == 2 ? (theLaunchIndex.x < theLaunchDim.x * 0.5) ? sysData.pane_a_flags : sysData.pane_b_flags
                                                                              : sysData.num_panes == 3   ? (theLaunchIndex.x < theLaunchDim.x * 0.33) ? sysData.pane_a_flags : (theLaunchIndex.x < theLaunchDim.x * 0.67) ? sysData.pane_b_flags
                                                                                                                                                                                                                          : sysData.pane_c_flags
                                                                                                         : sysData.pane_c_flags;

    if (pane_flags.do_reference)
    {
        prd.do_reference = true;
        prd.num_ris_samples = 0;
        prd.do_temporal_resampling = false;
        prd.do_spatial_resampling = false;
    }
    else
    {
        prd.do_reference = false;
        prd.num_ris_samples = pane_flags.ris_samples;
        prd.do_temporal_resampling = pane_flags.do_temporal_reuse;
        prd.do_spatial_resampling = pane_flags.do_spatial_reuse;
    }

    bool do_ris = prd.num_ris_samples > 0;
    // printf("CHECK GUARDS: do_ref = %i, do_ris = %i, do_temporal = %i, do_spatial = %i\n", prd.do_reference, prd.num_ris_samples, prd.do_temporal_resampling, prd.do_spatial_resampling);

    prd.launch_linear_index = lidx_ris;

    // clear out previous frame's temp buffer
    if (prd.do_temporal_resampling)
    {
        temp_reservoir_buffer[index] = Reservoir({0, 0, 0, 0});
    }
    float3 nearest_hit_current = make_float3(0.0);

    // ########################
    // HANDLE RIS LOGIC
    // ########################
    if (sysData.cur_iter != sysData.spp) {
        if (do_ris) {
            ris_output_reservoir_buffer[lidx_ris] = Reservoir({0, 0, 0, 0});
        }
        radiance = integrator(prd, index);
        // printf("AFTER INTEGRATION, radiance = %f, do_ris_resampling = %i \n", length(radiance), do_ris);

        if (do_ris) {
            nearest_hit_current = ris_output_reservoir_buffer[lidx_ris].nearest_hit;

            if (prd.do_spatial_resampling) {
                if (sysData.cur_iter == 0){
                    radiance += prd.radiance_first_hit;
                }
            }
            else {
                radiance += prd.radiance_first_hit;
            }
        }
    }

    // ########################
    //  HANDLE TEMPORAL LOGIC
    // ########################
    if (prd.do_temporal_resampling && !sysData.first_frame && sysData.cur_iter != sysData.spp) {
        Reservoir s = Reservoir({0, 0, 0, 0});

        Reservoir *current_pixel_prev_resevoir = &spatial_output_reservoir_buffer[lidx_ris]; // get current pixel's previous reservoir
        Reservoir *current_reservoir = &temp_reservoir_buffer[index];                        // choose current reservoir
        LightSample *y1 = &current_reservoir->y;

        updateReservoir(
            &s,
            y1,
            length(y1->radiance_over_pdf) * y1->pdf * current_reservoir->W * current_reservoir->M,
            &prd.seed
        );
        int2 current_pixel_prev_coord = pixel_from_world_coord(screen, ray, current_pixel_prev_resevoir->nearest_hit);
        int2 current_pixel_curr_coord = pixel_from_world_coord(screen, ray, current_reservoir->nearest_hit);
        int offset_x = theLaunchIndex.x - current_pixel_prev_coord.x;
        int offset_y = theLaunchIndex.y - current_pixel_prev_coord.y;
        int prev_coord_x = theLaunchIndex.x + offset_x;
        int prev_coord_y = theLaunchIndex.y + offset_y;

        bool prev_coord_offscreen = false;
        if (prev_coord_x < 0 || prev_coord_y < 0)
            prev_coord_offscreen = true;
        else if (prev_coord_x >= theLaunchDim.x || prev_coord_y >= theLaunchDim.y)
            prev_coord_offscreen = true;

        bool prev_coord_no_hit = true;
        if (
            current_reservoir->nearest_hit.x != 0.f &&
            current_reservoir->nearest_hit.y != 0.f &&
            current_reservoir->nearest_hit.z != 0.f
        ){
            prev_coord_no_hit = false;
        }

        bool prev_too_far = sqrt((double)(offset_x * offset_x + offset_y * offset_y)) > 30.f;

        if (!prev_coord_offscreen && !prev_coord_no_hit && !prev_too_far) {
            // select previous frame's reservoir and combine it
            // and only combine if you actually hit something (empty reservoir bad!)
            int prev_index =
                theLaunchDim.x * theLaunchDim.y * (sysData.cur_iter) +
                prev_coord_y * theLaunchDim.x + prev_coord_x; // TODO: how to calculate motion vector??

            Reservoir *prev_frame_reservoir = &spatial_output_reservoir_buffer[prev_index];

            LightSample *y2 = &prev_frame_reservoir->y;
            if (prev_frame_reservoir->M >= current_reservoir->M){
                prev_frame_reservoir->M = current_reservoir->M;
            }

            updateReservoir(
                &s,
                y2,
                length(y2->radiance_over_pdf) * y2->pdf * prev_frame_reservoir->W * prev_frame_reservoir->M,
                &prd.seed);

            s.M = current_reservoir->M + prev_frame_reservoir->M;
            s.W =
                (1.0f / (length(s.y.radiance_over_pdf) * s.y.pdf)) * // 1 / p_hat
                (1.0f / s.M) *
                s.w_sum;
            if (isnan(s.W) || s.M == 0.f) s.W = 0;
        
            s.nearest_hit = current_reservoir->nearest_hit;
            // s.y.throughput = y1->throughput;
            // s.y.bxdf = y1->bxdf;
            // s.y.weightMIS = y1->weightMIS;

            ris_output_reservoir_buffer[lidx_ris] = s;
        } else {
            ris_output_reservoir_buffer[lidx_ris] = *current_reservoir;
        }
    }

    // ########################
    // HANDLE SPATIAL LOGIC
    // ########################
    if (prd.do_spatial_resampling && sysData.cur_iter != 0){
        Reservoir updated_reservoir = ris_output_reservoir_buffer[lidx_spatial];
        float3 nearest_hit_current = updated_reservoir.nearest_hit;
        float3 current_throughput = updated_reservoir.y.throughput;
        float3 current_bxdf = updated_reservoir.y.bxdf;
        float current_weightMIS = updated_reservoir.y.weightMIS;

        if (updated_reservoir.W != 0){
            int k = 5;
            int radius = 30;
            int num_k_sampled = 0;
            int total_M = updated_reservoir.M;

            while (num_k_sampled < k){
                float2 sample = (rng2(prd.seed) - 0.5f) * radius * 2.0f;
                float squared_dist = sample.x * sample.x + sample.y * sample.y;
                if (squared_dist > radius * radius)
                    continue;

                int _x = (int)sample.x + theLaunchIndex.x;
                int _y = (int)sample.y + theLaunchIndex.y;
                if (_x < 0 || _x >= theLaunchDim.x)
                    continue;
                if (_y < 0 || _y >= theLaunchDim.y)
                    continue;
                if (_x == theLaunchIndex.x && _y == theLaunchIndex.y)
                    continue;

                unsigned int neighbor_index =
                    theLaunchDim.x * theLaunchDim.y * (sysData.cur_iter - 1) +
                    _y * theLaunchDim.x + _x;
                Reservoir *neighbor_reservoir = &ris_output_reservoir_buffer[neighbor_index];
                LightSample *y = &neighbor_reservoir->y;

                updateReservoir(
                    &updated_reservoir,
                    y,
                    length(y->radiance_over_pdf) * y->pdf * neighbor_reservoir->W * neighbor_reservoir->M,
                    &prd.seed);
                total_M += neighbor_reservoir->M;

                num_k_sampled += 1;
            }

            LightSample y = updated_reservoir.y;
            updated_reservoir.M = total_M;
            updated_reservoir.W =
                (1.0f / (length(y.radiance_over_pdf) * y.pdf)) * // 1 / p_hat
                (1.0f / updated_reservoir.M) *
                updated_reservoir.w_sum;
            updated_reservoir.nearest_hit = nearest_hit_current;

            updated_reservoir.y.bxdf = current_bxdf;
            updated_reservoir.y.throughput = current_throughput;
            updated_reservoir.y.weightMIS = current_weightMIS;

            spatial_output_reservoir_buffer[lidx_spatial] = updated_reservoir;
            radiance += current_throughput * current_bxdf * 
                y.radiance_over_pdf * y.pdf * // issue with using this pdf...
                updated_reservoir.W * sysData.numLights * current_weightMIS;
            // radiance += prd.radiance_first_hit;
        } else {
            radiance += prd.radiance_first_hit;
        }
    }

#if USE_DEBUG_EXCEPTIONS
    // DEBUG Highlight numerical errors.
    if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
    {
        radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
    }
    else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
    {
        radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
    }
    else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
    {
        radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
    }
#else
    // NaN values will never go away. Filter them out before they can arrive in the output buffer.
    // This only has an effect if the debug coloring above is off!
    if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
    {

#if USE_FP32_OUTPUT

        float4 *buffer = reinterpret_cast<float4 *>(sysData.outputBuffer);

#if USE_TIME_VIEW
        clock_t clockEnd = clock();
        const float alpha = (clockEnd - clockBegin) * sysData.clockScale;

        float4 result = make_float4(radiance, alpha);

        if (0 < sysData.cur_iter)
        {
            const float4 dst = buffer[index]; // RGBA32F

            result = lerp(dst, result, 1.0f / float(sysData.cur_iter + 1)); // Accumulate the alpha as well.
        }
        buffer[index] = result;
#else  // if !USE_TIME_VIEW
        if (sysData.iterationIndex < sysData.spp)
        { // FIXME
            if (0 < sysData.iterationIndex)
            {
                const float4 dst = buffer[index]; // RGBA32F

                radiance = lerp(make_float3(dst), radiance, 1.0f / float(sysData.iterationIndex + 1)); // Only accumulate the radiance, alpha stays 1.0f.
            }
            buffer[index] = make_float4(radiance, 1.0f);
        }
#endif // USE_TIME_VIEW

#else // if !USE_FP32_OUPUT

        Half4 *buffer = reinterpret_cast<Half4 *>(sysData.outputBuffer);

#if USE_TIME_VIEW
        clock_t clockEnd = clock();
        float alpha = (clockEnd - clockBegin) * sysData.clockScale;

        if (0 < sysData.cur_iter)
        {
            const float t = 1.0f / float(sysData.cur_iter + 1);

            const Half4 dst = buffer[index]; // RGBA16F

            radiance.x = lerp(__half2float(dst.x), radiance.x, t);
            radiance.y = lerp(__half2float(dst.y), radiance.y, t);
            radiance.z = lerp(__half2float(dst.z), radiance.z, t);
            alpha = lerp(__half2float(dst.z), alpha, t);
        }
        buffer[index] = make_Half4(radiance, alpha);
#else  // if !USE_TIME_VIEW
        if (0 < sysData.cur_iter)
        {
            const float t = 1.0f / float(sysData.cur_iter + 1);

            const Half4 dst = buffer[index]; // RGBA16F

            radiance.x = lerp(__half2float(dst.x), radiance.x, t);
            radiance.y = lerp(__half2float(dst.y), radiance.y, t);
            radiance.z = lerp(__half2float(dst.z), radiance.z, t);
        }
        buffer[index] = make_Half4(radiance, 1.0f);
#endif // USE_TIME_VIEW

#endif // USE_FP32_OUTPUT
    }
}
